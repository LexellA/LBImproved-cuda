#include <hip/hip_runtime.h>

#include "NearestNeighbor.h"
#include "dtw.h"
#include "LB_Keogh.h" 
#include "Envelope.h"



double LB_Keogh::test(const double* candidate)
{
    ++lb_keogh;

    hipStreamBeginCapture(mStream, hipStreamCaptureModeGlobal);

    // 将数据从CPU复制到GPU
    hipMemcpyAsync(d_candidate, candidate, size * sizeof(double),
               hipMemcpyHostToDevice, mStream);

    // 调用computeErrorKernel，计算出每个点的误差errors
    int threadsPerBlock = BLOCK_SZ;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    computeErrorKernel<<<blocksPerGrid, threadsPerBlock, 0, mStream>>>(
        U_K, L_K, d_candidate, d_errors, size);

    // 调用reduceKernel
    hipMemsetAsync(d_result, 0, sizeof(double), mStream);
    reduceKernel<<<blocksPerGrid, threadsPerBlock, 0, mStream>>>(
        d_errors, d_result, size);

    // 从GPU复制最终结果回CPU
    double error = 0;
    hipMemcpyAsync(&error, d_result, sizeof(double), hipMemcpyDeviceToHost,
                    mStream);

    hipStreamEndCapture(mStream, &mGraph);
    hipGraphInstantiate(&mGraphExec, mGraph, NULL, NULL, 0);
    hipGraphLaunch(mGraphExec, mStream);
    hipStreamSynchronize(mStream);
    hipGraphDestroy(mGraph);
    hipGraphExecDestroy(mGraphExec);

    // Continue with the rest of the test function
    if (error < bestsofar)
    {
      ++full_dtw;
      hipStreamBeginCapture(mStream, hipStreamCaptureModeGlobal);
      const double trueerror =
          mDTW.fastdynamic(V_K, d_candidate, mStream, mGraph, mGraphExec);
      if (trueerror < bestsofar) bestsofar = trueerror;
    }

    hipStreamSynchronize(mStream); 

    return bestsofar;
}


double LB_Keogh::getLowestCost() { return bestsofar; }

LB_Keogh::LB_Keogh(double* v, unsigned int v_size, unsigned int constraint)
    : NearestNeighbor(v, v_size, constraint),
      size(v_size),
      lb_keogh(0),
      full_dtw(0),
      bestsofar(dtw::INF),
      mGraphExec(NULL){

    hipStreamCreate(&mStream);

    hipMallocAsync(&V_K, size * sizeof(double), mStream);
    hipMallocAsync(&U_K, size * sizeof(double), mStream);
    hipMallocAsync(&L_K, size * sizeof(double), mStream);
    hipMallocAsync(&d_candidate, size * sizeof(double), mStream);
    hipMallocAsync(&d_errors, size * sizeof(double), mStream);
    hipMallocAsync(&d_result, sizeof(double), mStream);

    hipMemcpyAsync(V_K, v, size * sizeof(double), hipMemcpyHostToDevice,
                   mStream);

    Envelope envelope(V_K, U_K, L_K, size, constraint);
    envelope.compute(mStream);
}

LB_Keogh::~LB_Keogh()
{
  hipFreeAsync(V_K, mStream);
  hipFreeAsync(U_K, mStream);
  hipFreeAsync(L_K, mStream);

  hipFreeAsync(d_candidate, mStream);
  hipFreeAsync(d_errors, mStream);
  hipFreeAsync(d_result, mStream);
  hipStreamDestroy(mStream);
}