#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <chrono>

#include "NearestNeighbor.h"
#include "dtw.h"
#include "LB_Keogh.h" 
#include "Envelope.h"

__global__ void computeErrorKernel(const double *U, const double *L, const double *candidate, double *errors, unsigned int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size)
        return;

    double temp = candidate[i];
    double upper = U[i];
    double lower = L[i];

    // 避免分支，通过数学运算代替if-else
    errors[i] = max(0.0, temp - upper) + max(0.0, lower - temp);
}

__global__ void reduceKernel(double *input, double *output, unsigned int n)
{
    extern __shared__ double sdata[];

    // 每个线程负责读取一个元素到共享内存
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        sdata[tid] = input[i];
    else
        sdata[tid] = 0;
    __syncthreads();

    // 进行并行规约，每一步都将活动线程数减半
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // 将每个block的规约结果写回全局内存
    if (tid == 0)
        output[blockIdx.x] = sdata[0];
}

double LB_Keogh::test_kernel(const double* candidate)
{


    ++lb_keogh;

    // 分配device内存
    double  *d_candidate, *d_errors, *d_result;
    // hipMalloc(&d_V, size * sizeof(double));
    hipMalloc(&d_candidate, size * sizeof(double));
    hipMalloc(&d_errors, size * sizeof(double));

    // 将数据从CPU复制到GPU
    hipMemcpy(d_candidate, candidate, size * sizeof(double), hipMemcpyHostToDevice);

    // 调用computeErrorKernel，计算出每个点的误差errors
    int threadsPerBlock = 512;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    // 分配device内存用于存储每个block的规约结果
    hipMalloc(&d_result, blocksPerGrid * sizeof(double));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // auto start = std::chrono::high_resolution_clock::now();

    computeErrorKernel<<<blocksPerGrid, threadsPerBlock>>>(U_K, L_K, d_candidate, d_errors, size);

    // hipDeviceSynchronize();
    // auto end = std::chrono::high_resolution_clock::now();
    // auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

    // std::cout << "Time (computeErrorKernel): " << duration << " us" << std::endl;

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time (test_kernel): " << milliseconds << " ms" << std::endl;


    /**
     * 规约计算errors
     */ 


    // 每个block的共享内存大小
    int sharedMemSize = threadsPerBlock * sizeof(double); 
    // 调用reduceKernel
    reduceKernel<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_errors, d_result, size);

    // 从GPU复制最终结果回CPU
    double *h_result = new double[blocksPerGrid]; 
    hipMemcpy(h_result, d_result, blocksPerGrid * sizeof(double), hipMemcpyDeviceToHost);

    // 在CPU上完成最终的规约
    double error = 0.0;
    for (int i = 0; i < blocksPerGrid; i++)
    {
        error += h_result[i];
    }



    // Continue with the rest of the test function
    if (error < bestsofar)
    {
        ++full_dtw;
        const double trueerror = mDTW.fastdynamic(V_K, d_candidate);
        if (trueerror < bestsofar)
            bestsofar = trueerror;
    }

    // Free device memory
    hipFree(d_candidate);
    hipFree(d_errors);
    hipFree(d_result);

    return bestsofar;
}

double LB_Keogh::test(const double* candidate)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // auto start = std::chrono::high_resolution_clock::now();


    ++lb_keogh;
    double error(0.0);
    for (uint i = 0; i < size; ++i)
    {
        if (candidate[i] > U[i])
            error += candidate[i] - U[i];
        else if (candidate[i] < L[i])
            error += L[i] - candidate[i];
    }

    // auto end = std::chrono::high_resolution_clock::now();
    // auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
    // std::cout << "Time (test): " << duration << " us" << std::endl;

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time (test): " << milliseconds << " ms" << std::endl;


    if (error < bestsofar)
    {
        double *d_candidate;
        hipMalloc(&d_candidate, size * sizeof(double));
        hipMemcpy(d_candidate, candidate, size * sizeof(double), hipMemcpyHostToDevice);
        ++full_dtw;
        const double trueerror =
            mDTW.fastdynamic(V_K, d_candidate);
        if (trueerror < bestsofar)
            bestsofar = trueerror;

        hipFree(d_candidate);
    }
    return bestsofar;
}

double LB_Keogh::getLowestCost() { return bestsofar; }

LB_Keogh::LB_Keogh(double* v, unsigned int v_size, unsigned int constraint)
    : NearestNeighbor(v, v_size, constraint), size(v_size), lb_keogh(0), full_dtw(0), bestsofar(dtw::INF)
{
    V = v;
    U = new double[size];
    L = new double[size];
    hipMalloc(&V_K, size * sizeof(double));
    hipMalloc(&U_K, size * sizeof(double));
    hipMalloc(&L_K, size * sizeof(double));

    hipMemcpy(V_K, V, size * sizeof(double), hipMemcpyHostToDevice);

    Envelope envelope(V_K, U_K, L_K, size, mConstraint);
    envelope.compute();

    hipMemcpy(U, U_K, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(L, L_K, size * sizeof(double), hipMemcpyDeviceToHost);


    // for(uint i = 0; i < 100; ++i)
    // {
    //     std::cout << U[i] << " ";
    // }
    // for(uint i = 0; i < 100; ++i)
    // {
    //     std::cout << L[i] << " ";
    // }
}

LB_Keogh::~LB_Keogh()
{
    hipFree(V_K);
    hipFree(U_K);
    hipFree(L_K);
}