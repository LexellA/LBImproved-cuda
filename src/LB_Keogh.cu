#include <hip/hip_runtime.h>

#include "NearestNeighbor.h"
#include "dtw.h"
#include "LB_Keogh.h" 
#include "Envelope.h"



double LB_Keogh::test(const double* candidate)
{
    ++lb_keogh;

    // 将数据从CPU复制到GPU
    hipMemcpy(d_candidate, candidate, size * sizeof(double), hipMemcpyHostToDevice);

    // 调用computeErrorKernel，计算出每个点的误差errors
    int threadsPerBlock = BLOCK_SZ;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;


    computeErrorKernel<<<blocksPerGrid, threadsPerBlock>>>(
        U_K, L_K, d_candidate, d_errors, size);


    // 调用reduceKernel
    hipMemset(d_result, 0, sizeof(double));
    reduceKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_errors, d_result, size);

    // 从GPU复制最终结果回CPU
    double error = 0;
    hipMemcpy(&error, d_result, sizeof(double), hipMemcpyDeviceToHost);
    std::cout<<"error: "<<error<<std::endl;

    // Continue with the rest of the test function
    if (error < bestsofar)
    {
        ++full_dtw;
        const double trueerror = mDTW.fastdynamic(V_K, d_candidate);
        if (trueerror < bestsofar)
            bestsofar = trueerror;
    }

    return bestsofar;
}


double LB_Keogh::getLowestCost() { return bestsofar; }

LB_Keogh::LB_Keogh(double* v, unsigned int v_size, unsigned int constraint)
    : NearestNeighbor(v, v_size, constraint), size(v_size), lb_keogh(0), full_dtw(0), bestsofar(dtw::INF)
{
    hipMalloc(&V_K, size * sizeof(double));
    hipMalloc(&U_K, size * sizeof(double));
    hipMalloc(&L_K, size * sizeof(double));
    hipMalloc(&d_candidate, size * sizeof(double));
    hipMalloc(&d_errors, size * sizeof(double));
    hipMalloc(&d_result, sizeof(double));

    hipMemcpy(V_K, v, size * sizeof(double), hipMemcpyHostToDevice);

    Envelope envelope(V_K, U_K, L_K, size, constraint);
    envelope.compute();
}

LB_Keogh::~LB_Keogh()
{
    hipFree(V_K);
    hipFree(U_K);
    hipFree(L_K);

    hipFree(d_candidate);
    hipFree(d_errors);
    hipFree(d_result);
}