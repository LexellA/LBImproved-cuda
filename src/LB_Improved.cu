#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <chrono>

#include "NearestNeighbor.h"
#include "dtw.h"
#include "LB_Improved.h"
#include "LB_Keogh.h"
#include "Envelope.h"

__global__ void computeErrorKernelBuffer(const double *U, const double *L, const double *candidate, double *errors, double *buffer, unsigned int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    double temp = candidate[i];
    double upper = U[i];
    double lower = L[i];
    double error = 0, r_buffer = 0;
    if (i < size)
    {
        if (temp > upper)
        {
            error = temp - upper;
            r_buffer = upper;
        }
        else if (temp < lower)
        {
            error = lower - temp;
            r_buffer = lower;
        }
        else
        {
            error = 0;
        }
    }
    errors[i] = error;
    buffer[i] = r_buffer;
}



/**
 * 规约计算数组和
 */
uint compute_sum(double *d_array, uint size)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    int sharedMemSize = threadsPerBlock * sizeof(double);

    double *d_result;
    hipMalloc(&d_result, blocksPerGrid * sizeof(double));

    reduceKernel<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_array, d_result, size);
    // 从GPU复制最终结果回CPU
    double *h_result = new double[blocksPerGrid];
    hipMemcpy(h_result, d_result, blocksPerGrid * sizeof(double), hipMemcpyDeviceToHost);

    // 在CPU上完成最终的规约
    double sum = 0.0;
    for (int i = 0; i < blocksPerGrid; i++)
    {
        sum += h_result[i];
    }

    // Free device memory
    hipFree(d_result);
    free(h_result);

    return sum;
}

double LB_Improved::test_kernel(const double *candidate)
{
    ++lb_keogh;

    // 分配device内存
    double *d_candidate, *d_errors, *d_buffer;
    // hipMalloc(&d_V, size * sizeof(double));
    hipMalloc(&d_candidate, size * sizeof(double));
    hipMalloc(&d_errors, size * sizeof(double));
    hipMalloc(&d_buffer, size * sizeof(double));

    // 将数据从CPU复制到GPU
    hipMemcpy(d_candidate, candidate, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_buffer, candidate, size * sizeof(double), hipMemcpyHostToDevice);

    // 第二次LB_Keogh
    double *d_U2, *d_L2;
    hipMalloc(&d_U2, size * sizeof(double));
    hipMalloc(&d_L2, size * sizeof(double));

    // 调用computeErrorKernel，计算出每个点的误差errors
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // auto start = std::chrono::high_resolution_clock::now();

    computeErrorKernelBuffer<<<blocksPerGrid, threadsPerBlock>>>(U_K, L_K, d_candidate, d_errors, d_buffer, size);

    // hipDeviceSynchronize();
    // auto end = std::chrono::high_resolution_clock::now();
    // auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

    // std::cout << "Time (computeErrorKernel): " << duration << " us" << std::endl;



    double error = compute_sum(d_errors, size);

    // Continue with the rest of the test function
    if (error < bestsofar)
    {
        Envelope envelope(d_buffer, d_U2, d_L2, size, mConstraint);
        envelope.compute();

        computeErrorKernel<<<blocksPerGrid, threadsPerBlock>>>(d_U2, d_L2, V_K, d_errors, size);

        double error2 = compute_sum(d_errors, size);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        std::cout << "Time (test_kernel): " << milliseconds << " ms" << std::endl;

        if (error2 < bestsofar)
        {
            ++full_dtw;
            const double trueerror = mDTW.fastdynamic(V_K, d_candidate);
            if (trueerror < bestsofar)
                bestsofar = trueerror;
        }
    }

    // Free device memory
    hipFree(d_U2);
    hipFree(d_L2);

    hipFree(d_candidate);
    hipFree(d_errors);
    hipFree(d_buffer);

    return bestsofar;
}

double LB_Improved::test(const double *candidate)
{
        double *d_U2, *d_L2,  *U2, *L2;
        double * d_buffer;
        U2 = new double[size];
        L2 = new double[size];
        hipMalloc(&d_U2, size * sizeof(double));
        hipMalloc(&d_L2, size * sizeof(double));
        hipMalloc(&d_buffer, size * sizeof(double));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    ++lb_keogh;
    double error(0.0);
    double *buffer = new double[size];
    for (uint i = 0; i < size; ++i)
    {
        const double &cdi(candidate[i]);
        if (cdi > U[i])
        {
            error += cdi - (buffer[i] = U[i]);
        }
        else if (cdi < L[i])
        {
            error += (buffer[i] = L[i]) - cdi;
        }
        else
            buffer[i] = cdi;
        if (error > bestsofar)
            return bestsofar;
    }

    if (error < bestsofar)
    {

        hipMemcpy(d_buffer, buffer, size * sizeof(double), hipMemcpyHostToDevice);

        Envelope envelope(d_buffer, d_U2, d_L2, size, mConstraint);

        envelope.compute();

        hipMemcpy(U2, d_U2, size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(L2, d_L2, size * sizeof(double), hipMemcpyDeviceToHost);

        for (uint i = 0; i < size; ++i)
        {
            if (V[i] > U2[i])
            {
                error += V[i] - U2[i];
            }
            else if (V[i] < L2[i])
            {
                error += L2[i] - V[i];
            }
            if (error > bestsofar)
                return bestsofar;
        }

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        std::cout << "Time (test): " << milliseconds << " ms" << std::endl;

        double *d_candidate;
        hipMalloc(&d_candidate, size * sizeof(double));
        hipMemcpy(d_candidate, candidate, size * sizeof(double), hipMemcpyHostToDevice);

        if (error < bestsofar)
        {
            ++full_dtw;
            const double trueerror =
                mDTW.fastdynamic(V_K, d_candidate); //,mConstraint,1);
            if (trueerror < bestsofar)
                bestsofar = trueerror;
        }

        
    }
        hipFree(d_U2);
        hipFree(d_L2);
        hipFree(d_buffer);
        free(U2);
        free(L2);
    return bestsofar;
}

double LB_Improved::getLowestCost() { return bestsofar; }

LB_Improved::LB_Improved(double *v, unsigned int v_size, unsigned int constraint)
    : NearestNeighbor(v, v_size, constraint), size(v_size), lb_keogh(0), full_dtw(0), bestsofar(dtw::INF)
{
    V = v;
    U = new double[size];
    L = new double[size];
    hipMalloc(&V_K, size * sizeof(double));
    hipMalloc(&U_K, size * sizeof(double));
    hipMalloc(&L_K, size * sizeof(double));

    hipMemcpy(V_K, V, size * sizeof(double), hipMemcpyHostToDevice);

    Envelope envelope(V_K, U_K, L_K, size, mConstraint);
    envelope.compute();

    hipMemcpy(U, U_K, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(L, L_K, size * sizeof(double), hipMemcpyDeviceToHost);
}

LB_Improved::~LB_Improved()
{
    hipFree(V_K);
    hipFree(U_K);
    hipFree(L_K);
    free(U);
    free(L);
}

// double LB_Improved::fastdynamic111(const double *v, const double *w)
// {


//     double **mGamma;
//     int mN = size;
//     mGamma = new double *[mN];
//     for (int i = 0; i < mN; ++i)
//         mGamma[i] = new double[mN];

//     double Best(dtw::INF);
//     for (int i = 0; i < mN; ++i)
//     {
//         for (int j = max(0, i - mConstraint); j < min(mN, i + mConstraint + 1);
//              ++j)
//         {
//             Best = dtw::INF;
//             if (i > 0)
//                 Best = mGamma[i - 1][j];
//             if (j > 0)
//                 Best = min(Best, mGamma[i][j - 1]);
//             if ((i > 0) && (j > 0))
//                 Best = min(Best, mGamma[i - 1][j - 1]);
//             if ((i == 0) && (j == 0))
//                 mGamma[i][j] = fabs(v[i] - w[j]);
//             else
//                 mGamma[i][j] = Best + fabs(v[i] - w[j]);
//         }
//     }
//     return mGamma[mN - 1][mN - 1];
// }
