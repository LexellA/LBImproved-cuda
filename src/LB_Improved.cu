#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "NearestNeighbor.h"
#include "dtw.h"
#include "LB_Improved.h"
#include "Envelope.h"


__global__ void computeErrorKernelBuffer(const double *U, const double *L,
                                         const double *candidate,
                                         double *errors, double *buffer,
                                         unsigned int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        double error = 0, r_buffer = 0;
        double temp = candidate[i];
        double upper = U[i];
        double lower = L[i];
        if (temp > upper)
        {
            error = temp - upper;
            r_buffer = upper;
        }
        else if (temp < lower)
        {
            error = lower - temp;
            r_buffer = lower;
        }
        else
        {
          error = 0;
            r_buffer = temp;
        }
        errors[i] = error;
        buffer[i] = r_buffer;
    }
}



/**
 * 规约计算数组和
 */
double LB_Improved::compute_sum(double *d_array, uint size) {
  int threadsPerBlock = BLOCK_SZ;
  int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;


  hipMemsetAsync(d_result, 0, sizeof(double), mStream);
  reduceKernel<<<blocksPerGrid, threadsPerBlock,0, mStream>>>(
      d_array, d_result, size);
  // 从GPU复制最终结果回CPU
  double h_result;
  hipMemcpyAsync(&h_result, d_result, sizeof(double), hipMemcpyDeviceToHost,
                  mStream);
  
  hipStreamEndCapture(mStream, &mGraph);
  hipGraphInstantiate(&mGraphExec, mGraph, NULL, NULL, 0);
  hipGraphLaunch(mGraphExec, mStream);
  hipStreamSynchronize(mStream);
  hipGraphExecDestroy(mGraphExec);
  hipGraphDestroy(mGraph);
  return h_result;
}

double LB_Improved::test(const double *candidate)
{
    ++lb_keogh;
    // 将数据从CPU复制到GPU
    hipStreamBeginCapture(mStream, hipStreamCaptureModeGlobal);
    
    hipMemcpyAsync(d_candidate, candidate, size * sizeof(double),
                    hipMemcpyHostToDevice, mStream);
    

    // 调用computeErrorKernel，计算出每个点的误差errors
    int threadsPerBlock = LB_Improved::BLOCK_SZ;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    computeErrorKernelBuffer<<<blocksPerGrid, threadsPerBlock, 0, mStream>>>(
        U_K, L_K, d_candidate, d_errors, d_buffer, size);

    double error = compute_sum(d_errors, size);

    

    // Continue with the rest of the test function
    if (error < bestsofar) {

      // 第二次LB_Keogh
        hipStreamBeginCapture(mStream, hipStreamCaptureModeGlobal);
        Envelope envelope(d_buffer, d_U2, d_L2, size, mConstraint);
        envelope.compute(mStream);

        computeErrorKernel<<<blocksPerGrid, threadsPerBlock, 0, mStream>>>(d_U2, d_L2, V_K, d_errors, size);

        double error2 = compute_sum(d_errors, size);

        error2 += error;

        if (error2 < bestsofar)
        {
          ++full_dtw;
          hipStreamBeginCapture(mStream, hipStreamCaptureModeGlobal);
          const double trueerror = mDTW.fastdynamic(V_K, d_candidate, mStream, mGraph, mGraphExec);

          if (trueerror < bestsofar) bestsofar = trueerror;
        }
    }

    hipStreamSynchronize(mStream);

    return bestsofar;
}

double LB_Improved::getLowestCost() { return bestsofar; }

LB_Improved::LB_Improved(double *v, unsigned int v_size,
                         unsigned int constraint)
    : NearestNeighbor(v, v_size, constraint),
      size(v_size),
      lb_keogh(0),
      full_dtw(0),
      bestsofar(dtw::INF),
      mConstraint(constraint),
      mGraphExec(NULL)
      {

    hipStreamCreate(&mStream);

    hipMallocAsync(&V_K, size * sizeof(double), mStream);
    hipMallocAsync(&U_K, size * sizeof(double), mStream);
    hipMallocAsync(&L_K, size * sizeof(double), mStream);

    hipMallocAsync(&d_candidate, size * sizeof(double), mStream);
    hipMallocAsync(&d_errors, size * sizeof(double), mStream);
    hipMallocAsync(&d_buffer, size * sizeof(double), mStream);
    hipMallocAsync(&d_U2, size * sizeof(double), mStream);
    hipMallocAsync(&d_L2, size * sizeof(double), mStream);

    hipMallocAsync(&d_result, sizeof(double), mStream);

    hipMemcpyAsync(V_K, v, size * sizeof(double), hipMemcpyHostToDevice,
                    mStream);

    Envelope envelope(V_K, U_K, L_K, size, mConstraint);
    envelope.compute(mStream);
    hipStreamSynchronize(mStream);
}

LB_Improved::~LB_Improved()
{
  hipFreeAsync(V_K, mStream);
  hipFreeAsync(U_K, mStream);
  hipFreeAsync(L_K, mStream);

  hipFreeAsync(d_candidate, mStream);
  hipFreeAsync(d_errors, mStream);
  hipFreeAsync(d_buffer, mStream);

  hipFreeAsync(d_U2, mStream);
  hipFreeAsync(d_L2, mStream);

  hipFreeAsync(d_result, mStream);
  hipStreamDestroy(mStream);
}