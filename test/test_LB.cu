#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "dtw.h"
#include "LB_Keogh.h"

// Helper function to check CUDA error status
void checkCudaStatus(hipError_t status, const char *msg)
{
    if (status != hipSuccess)
    {
        std::cerr << "CUDA Error: " << msg << ": " << hipGetErrorString(status) << std::endl;
        exit(EXIT_FAILURE);
    }
}

// Function to generate random walk data
std::vector<double> getrandomwalk(uint size)
{
    std::vector<double> data(size);
    data[0] = 0.0;
    for (uint k = 1; k < size; ++k)
        data[k] = (1.0 * rand() / RAND_MAX) - 0.5 + data[k - 1];
    return data;
}

// Function to demo the LB_Keogh class with random walk data
void demo(uint size)
{
    std::cout << "Generating random walk and matching it with other random walks..." << std::endl;

    std::vector<double> target = getrandomwalk(size); // This is our target
    LB_Keogh filter(target, size / 10);               // Use DTW with a tolerance of 10% (size/10)
    double bestsofar = filter.getLowestCost();
    uint howmany = 5000;

    // Allocate CUDA events for original test function
    hipEvent_t start, stop;
    checkCudaStatus(hipEventCreate(&start), "Failed to create start event");
    checkCudaStatus(hipEventCreate(&stop), "Failed to create stop event");

    float totalMillisecondsTest = 0.0f;
    float totalMillisecondsTestKernel = 0.0f;

    for (uint i = 0; i < howmany; ++i)
    {
        std::vector<double> candidate = getrandomwalk(size);

        // Timing the original test function
        checkCudaStatus(hipEventRecord(start), "Failed to record start event");
        double newbest = filter.test(candidate);
        checkCudaStatus(hipEventRecord(stop), "Failed to record stop event");
        checkCudaStatus(hipEventSynchronize(stop), "Failed to synchronize stop event");
        float millisecondsTest = 0;
        checkCudaStatus(hipEventElapsedTime(&millisecondsTest, start, stop), "Failed to get elapsed time");
        totalMillisecondsTest += millisecondsTest;

        // Timing the test_kernel function
        checkCudaStatus(hipEventRecord(start), "Failed to record start event");
        double newbestKernel = filter.test_kernel(candidate);
        checkCudaStatus(hipEventRecord(stop), "Failed to record stop event");
        checkCudaStatus(hipEventSynchronize(stop), "Failed to synchronize stop event");
        float millisecondsTestKernel = 0;
        checkCudaStatus(hipEventElapsedTime(&millisecondsTestKernel, start, stop), "Failed to get elapsed time");
        totalMillisecondsTestKernel += millisecondsTestKernel;

        if (newbest < bestsofar || newbestKernel < bestsofar)
        {
            std::cout << "Found a new nearest neighbor, distance (L1 norm) = " << newbest << std::endl;
            bestsofar = newbest;
        }

        std::cout << "Iteration: " << i + 1 << ", Time (test): " << millisecondsTest << " ms, Time (test_kernel): " << millisecondsTestKernel << " ms" << std::endl;
    }

    std::cout << "Compared with " << howmany << " random walks, closest match is at a distance (L1 norm) of " << filter.getLowestCost() << std::endl;
    std::cout << "Average time (test): " << totalMillisecondsTest / howmany << " ms" << std::endl;
    std::cout << "Average time (test_kernel): " << totalMillisecondsTestKernel / howmany << " ms" << std::endl;

    // Clean up
    checkCudaStatus(hipEventDestroy(start), "Failed to destroy start event");
    checkCudaStatus(hipEventDestroy(stop), "Failed to destroy stop event");
}

int main()
{
    demo(10000);
    return 0;
}
