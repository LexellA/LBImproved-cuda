#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cassert>
#include <vector>
#include "LB_Keogh.h"
#include "dtw_origin.h"

// Helper function to check CUDA error status
void checkCudaStatus(hipError_t status, const char *msg)
{
    if (status != hipSuccess)
    {
        std::cerr << "CUDA Error: " << msg << ": " << hipGetErrorString(status) << std::endl;
        exit(EXIT_FAILURE);
    }
}

// Function to generate random walk data
void getrandomwalk(double *data, uint size)
{
    data[0] = 0.0;
    for (uint k = 1; k < size; ++k)
        data[k] = (1.0 * rand() / RAND_MAX) - 0.5 + data[k - 1];
}

// Function to demo the LB_Keogh class with random walk data
void demo(uint size)
{
    std::cout << "Generating random walk and matching it with other random walks..." << std::endl;

    double *target = new double[size];
    getrandomwalk(target, size);
    LB_Keogh filter(target, size,  size / 10);               // Use DTW with a tolerance of 10% (size/10)
    double bestsofar = filter.getLowestCost();
    uint howmany = 10;

    std::vector<double> target_origin(target, target + size);
    Origin::LB_Keogh filter_origin(target_origin, size / 10);        // Use the original DTW with a tolerance of 10% (size/10
    
    // Allocate CUDA events for original test function
    hipEvent_t start, stop;
    checkCudaStatus(hipEventCreate(&start), "Failed to create start event");
    checkCudaStatus(hipEventCreate(&stop), "Failed to create stop event");

    float totalMillisecondsTest = 0.0f;
    float totalMillisecondsTestKernel = 0.0f;

    for (uint i = 0; i < howmany; ++i)
    {
        double *candidate = new double[size];
        getrandomwalk(candidate, size);

        // Timing the test_kernel function
        checkCudaStatus(hipEventRecord(start), "Failed to record start event");
        double newbestKernel = filter.test(candidate);
        checkCudaStatus(hipEventRecord(stop), "Failed to record stop event");
        checkCudaStatus(hipEventSynchronize(stop), "Failed to synchronize stop event");
        float millisecondsTestKernel = 0;
        checkCudaStatus(hipEventElapsedTime(&millisecondsTestKernel, start, stop), "Failed to get elapsed time");
        totalMillisecondsTestKernel += millisecondsTestKernel;

        // Timing the original test function
        checkCudaStatus(hipEventRecord(start), "Failed to record start event");
        std::vector<double> candidate_origin(candidate, candidate + size);
        double newbest = filter_origin.test(candidate_origin);
        checkCudaStatus(hipEventRecord(stop), "Failed to record stop event");
        checkCudaStatus(hipEventSynchronize(stop), "Failed to synchronize stop event");
        float millisecondsTest = 0;
        checkCudaStatus(hipEventElapsedTime(&millisecondsTest, start, stop), "Failed to get elapsed time");
        totalMillisecondsTest += millisecondsTest;

        std::cout << i <<std::endl;
        assert(newbest == newbestKernel);

        if (newbest < bestsofar || newbestKernel < bestsofar)
        {
            // std::cout << "Found a new nearest neighbor, distance (L1 norm) = " << newbest << std::endl;
            bestsofar = newbest;
        }

        // std::cout << "Iteration: " << i + 1 << ", Time (test): " << millisecondsTest << " ms, Time (test_kernel): " << millisecondsTestKernel << " ms" << std::endl;
    }

    std::cout << "Compared with " << howmany << " random walks, closest match is at a distance (L1 norm) of " << filter.getLowestCost() << std::endl;
    std::cout << "Average time (test): " << totalMillisecondsTest / howmany << " ms" << std::endl;
    std::cout << "Average time (test_kernel): " << totalMillisecondsTestKernel / howmany << " ms" << std::endl;

    // Clean up
    checkCudaStatus(hipEventDestroy(start), "Failed to destroy start event");
    checkCudaStatus(hipEventDestroy(stop), "Failed to destroy stop event");
}

int main()
{
  demo(10);
  
    return 0;
}
